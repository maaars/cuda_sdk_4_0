/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
 
/*
* This sample demonstrates how use texture fetches in CUDA
*
* This sample takes an input PGM image (image_filename) and generates 
* an output PGM image (image_filename_out).  This CUDA kernel performs
* a simple 2D transform (rotation) on the texture coordinates (u,v).
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>
#include <shrQATest.h>

// includes, kernels
#include <simpleSurfaceWrite_kernel.cu>

char *image_filename = "lena_bw.pgm";
char *ref_filename   = "ref_rotated.pgm";
float angle = 0.5f;    // angle to rotate image by (in radians)

static char *sSDKname = "simpleSurfaceWrite";

#define MIN_EPSILON_ERROR 5e-3f


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

extern "C"
void computeGold( float* reference, float* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    bool bTestResult = true;
    shrQAStart(argc, argv);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
	int devID = cutilChooseCudaDevice(argc, argv);

    // get number of SMs on this GPU
    hipDeviceProp_t deviceProps;

	cutilSafeCall(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s] has %d Multi-Processors, SM %d.%d\n", deviceProps.name, deviceProps.multiProcessorCount, deviceProps.major, deviceProps.minor );

	if (deviceProps.major < 2) {
		printf("%s requires SM >= 2.0 for SurfaceWrites, exiting... \n", sSDKname);
		cutilDeviceReset();
        shrQAFinishExit(argc, (const char **)argv, QA_WAIVED);
	}

	// load image from disk
    float* h_data = NULL;
    unsigned int width, height;
    char* image_path = cutFindFilePath(image_filename, argv[0]);
    if (image_path == NULL) {
        printf("Unable to source image input file: %s\n", image_filename);
        shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
    }
    cutilCheckError( cutLoadPGMf(image_path, &h_data, &width, &height));

    unsigned int size = width * height * sizeof(float);
    printf("Loaded '%s', %d x %d pixels\n", image_filename, width, height);

    // load reference image from image (output)
    float *h_data_ref = (float*) malloc(size);
    char* ref_path = cutFindFilePath(ref_filename, argv[0]);
    if (ref_path == NULL) {
        printf("Unable to find reference image file: %s\n", ref_filename);
        shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
    }
    cutilCheckError( cutLoadPGMf(ref_path, &h_data_ref, &width, &height));

	// allocate device memory for result
    float* d_data = NULL;
    cutilSafeCall( hipMalloc( (void**) &d_data, size));

    // allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* cu_array;
    cutilSafeCall( hipMallocArray( &cu_array, &channelDesc, width, height, hipArraySurfaceLoadStore )); 

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

#if 1
    cutilSafeCall( hipMemcpy( d_data, h_data, size, hipMemcpyHostToDevice) );
    cutilSafeCall(cudaBindSurfaceToArray(output_surface, cu_array));

    surfaceWriteKernel<<< dimGrid, dimBlock >>>( d_data, width, height);
#else // this is what differs from the example simpleTexture
    cutilSafeCall( hipMemcpyToArray( cu_array, 0, 0, h_data, size, hipMemcpyHostToDevice));
#endif

    // set texture parameters
    tex.addressMode[0] = hipAddressModeWrap;
    tex.addressMode[1] = hipAddressModeWrap;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = true;    // access with normalized texture coordinates

    // Bind the array to the texture
    cutilSafeCall( hipBindTextureToArray( tex, cu_array, channelDesc));

    // warmup
    transformKernel<<< dimGrid, dimBlock, 0 >>>( d_data, width, height, angle);

    cutilSafeCall( cutilDeviceSynchronize() );
    unsigned int timer = 0;
    cutilCheckError( cutCreateTimer( &timer));
    cutilCheckError( cutStartTimer( timer));

    // execute the kernel
    transformKernel<<< dimGrid, dimBlock, 0 >>>( d_data, width, height, angle);

    // check if kernel execution generated an error
    cutilCheckMsg("Kernel execution failed");

    cutilSafeCall( cutilDeviceSynchronize() );
    cutilCheckError( cutStopTimer( timer));
    printf("Processing time: %f (ms)\n", cutGetTimerValue( timer));
    printf("%.2f Mpixels/sec\n", (width*height / (cutGetTimerValue( timer) / 1000.0f)) / 1e6);
    cutilCheckError( cutDeleteTimer( timer));

    // allocate mem for the result on host side
    float* h_odata = (float*) malloc( size);
    // copy result from device to host
    cutilSafeCall( hipMemcpy( h_odata, d_data, size, hipMemcpyDeviceToHost) );

    // write result to file
    char output_filename[1024];
    strcpy(output_filename, "output.pgm");
    cutilCheckError( cutSavePGMf( "output.pgm", h_odata, width, height) );
    printf("Wrote '%s'\n", output_filename);

    // write regression file if necessary
    if( cutCheckCmdLineFlag( argc, (const char**) argv, "regression"))
    {
        // write file for regression test
        cutilCheckError( cutWriteFilef( "./data/regression.dat", h_odata, width*height, 0.0));
    } 
    else 
    {
        // We need to reload the data from disk, because it is inverted upon output
        cutilCheckError( cutLoadPGMf(output_filename, &h_odata, &width, &height));

        printf("Comparing files\n");
        printf("\toutput:    <%s>\n", output_filename);
        printf("\treference: <%s>\n", ref_path);
        bTestResult = (bool)cutComparefe( h_odata, h_data_ref, width*height, MIN_EPSILON_ERROR );
    }

    cutilSafeCall(hipFree(d_data));
    cutilSafeCall(hipFreeArray(cu_array));
    cutFree(image_path);
    cutFree(ref_path);

    cutilDeviceReset();
    shrQAFinishExit(argc, (const char **)argv, (bTestResult ? QA_PASSED : QA_FAILED) );
}
