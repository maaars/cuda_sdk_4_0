#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <cutil_inline.h>
#include <shrQATest.h>
#include <hip/hip_runtime.h>

/* Add two vectors on the GPU */

__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < N)
    c[idx] = a[idx] + b[idx];
}

// Allocate generic memory with malloc() and pin it laster instead of using hipHostAlloc()
bool bPinGenericMemory = false;

// Macro to aligned up to the memory size in question
#define MEMORY_ALIGNMENT  4096
#define ALIGN_UP(x,size) ( ((size_t)x+(size-1))&(~(size-1)) )

int main(int argc, char **argv)
{
  int n, nelem, deviceCount;
  int idev = 0; // use default device 0
  char *device = NULL;
  unsigned int flags;
  size_t bytes;
  float *a, *b, *c;                      // Pinned memory allocated on the CPU
  float *a_UA, *b_UA, *c_UA;             // Non-4K Aligned Pinned memory on the CPU
  float *d_a, *d_b, *d_c;                // Device pointers for mapped memory  
  float errorNorm, refNorm, ref, diff;
  hipDeviceProp_t deviceProp;

  shrQAStart(argc, argv);

  if(cutCheckCmdLineFlag(argc, (const char **)argv, "help"))
  {
    printf("Usage:  simpleZeroCopy [OPTION]\n\n");
    printf("Options:\n");
    printf("  --device=[device #]  Specify the device to be used\n");
    printf("  --use_generic_memory (optional) use generic page-aligned for system memory\n");
    shrQAFinishExit(argc, (const char **)argv, QA_WAIVED);
  }

  /* Get the device selected by the user or default to 0, and then set it. */
  if(cutGetCmdLineArgumentstr(argc, (const char**)argv, "device", &device))
  {
    hipGetDeviceCount(&deviceCount);
    idev = atoi(device);
    if(idev >= deviceCount || idev < 0)
    {
      fprintf(stderr, "Device number %d is invalid, will use default CUDA device 0.\n", idev);
      idev = 0;
    }
  }
  
  if( cutCheckCmdLineFlag( argc, (const char **)argv, "use_generic_memory") ) 
  {
#if defined(__APPLE__) || defined(MACOSX)
    bPinGenericMemory = false;  // Generic Pinning of System Paged memory is not currently supported on Mac OSX 
#else
    bPinGenericMemory = true;
#endif
  }

  if (bPinGenericMemory) {
     printf("> Using Generic System Paged Memory (malloc)\n");
  } else {
     printf("> Using CUDA Host Allocated (hipHostAlloc)\n");
  }

  cutilSafeCall(hipSetDevice(idev));

  /* Verify the selected device supports mapped memory and set the device
     flags for mapping host memory. */

  cutilSafeCall(hipGetDeviceProperties(&deviceProp, idev));

#if CUDART_VERSION >= 2020
  if(!deviceProp.canMapHostMemory)
  {
    fprintf(stderr, "Device %d does not support mapping CPU host memory!\n", idev);
    cutilDeviceReset();	
    shrQAFinishExit(argc, (const char **)argv, QA_PASSED);
  }
  cutilSafeCall(hipSetDeviceFlags(hipDeviceMapHost));
#else
    fprintf(stderr, "CUDART version %d.%d does not support <hipDeviceProp_t.canMapHostMemory> field\n", , CUDART_VERSION/1000, (CUDART_VERSION%100)/10);
    cutilDeviceReset();	
    shrQAFinishExit(argc, (const char **)argv, QA_PASSED);
#endif

#if CUDART_VERSION < 4000
  if (bPinGenericMemory)
  {
    fprintf(stderr, "CUDART version %d.%d does not support <hipHostRegister> function\n", CUDART_VERSION/1000, (CUDART_VERSION%100)/10);
    cutilDeviceReset();	
    shrQAFinishExit(argc, (const char **)argv, QA_PASSED);
  }
#endif

  /* Allocate mapped CPU memory. */

  nelem = 1048576;
  bytes = nelem*sizeof(float);
  if (bPinGenericMemory)
  {
#if CUDART_VERSION >= 4000
    a_UA = (float *) malloc( bytes + MEMORY_ALIGNMENT );
    b_UA = (float *) malloc( bytes + MEMORY_ALIGNMENT );
    c_UA = (float *) malloc( bytes + MEMORY_ALIGNMENT );

    // We need to ensure memory is aligned to 4K (so we will need to padd memory accordingly)
    a = (float *) ALIGN_UP( a_UA, MEMORY_ALIGNMENT );
    b = (float *) ALIGN_UP( b_UA, MEMORY_ALIGNMENT );
    c = (float *) ALIGN_UP( c_UA, MEMORY_ALIGNMENT );

    cutilSafeCall(hipHostRegister(a, bytes, hipHostMallocMapped));
    cutilSafeCall(hipHostRegister(b, bytes, hipHostMallocMapped));
    cutilSafeCall(hipHostRegister(c, bytes, hipHostMallocMapped));
#endif
  }
  else
  {
#if CUDART_VERSION >= 2020
    flags = hipHostMallocMapped;
    cutilSafeCall(hipHostAlloc((void **)&a, bytes, flags));
    cutilSafeCall(hipHostAlloc((void **)&b, bytes, flags));
    cutilSafeCall(hipHostAlloc((void **)&c, bytes, flags));
#endif
  }

  /* Initialize the vectors. */

  for(n = 0; n < nelem; n++)
  {
    a[n] = rand() / (float)RAND_MAX;
    b[n] = rand() / (float)RAND_MAX;
  }

  /* Get the device pointers for the pinned CPU memory mapped into the GPU
     memory space. */

#if CUDART_VERSION >= 2020
  cutilSafeCall(hipHostGetDevicePointer((void **)&d_a, (void *)a, 0));
  cutilSafeCall(hipHostGetDevicePointer((void **)&d_b, (void *)b, 0));
  cutilSafeCall(hipHostGetDevicePointer((void **)&d_c, (void *)c, 0));
#endif

  /* Call the GPU kernel using the CPU pointers residing in CPU mapped memory. */ 
  printf("> vectorAddGPU kernel will add vectors using mapped CPU memory...\n");
  dim3 block(256);
  dim3 grid((unsigned int)ceil(nelem/(float)block.x));
  vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, nelem);  
  cutilSafeCall(cutilDeviceSynchronize());
  cutilCheckMsg("vectorAddGPU() execution failed");

  /* Compare the results */

  printf("> Checking the results from vectorAddGPU() ...\n");
  errorNorm = 0.f;
  refNorm = 0.f;
  for(n = 0; n < nelem; n++)
  {
    ref = a[n] + b[n];
    diff = c[n] - ref;
    errorNorm += diff*diff;
    refNorm += ref*ref;
  }
  errorNorm = (float)sqrt((double)errorNorm);
  refNorm = (float)sqrt((double)refNorm);

  /* Memory clean up */

  printf("> Releasing CPU memory...\n");
  if (bPinGenericMemory)
  {
#if CUDART_VERSION >= 4000
    cutilSafeCall(hipHostUnregister(a));
    cutilSafeCall(hipHostUnregister(b));
    cutilSafeCall(hipHostUnregister(c));
    free(a_UA);
    free(b_UA);
    free(c_UA);
#endif
  }
  else
  {
#if CUDART_VERSION >= 2020
    cutilSafeCall(hipHostFree(a));
    cutilSafeCall(hipHostFree(b));
    cutilSafeCall(hipHostFree(c));
#endif
  }

  cutilDeviceReset();	
  shrQAFinishExit(argc, (const char **)argv, (errorNorm/refNorm < 1.e-6f) ? QA_PASSED : QA_FAILED);
}
